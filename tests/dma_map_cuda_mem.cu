#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <bafs.h>

#include <hip/hip_runtime.h>

#define PAGE_SIZE 65536

#define ALIGN_SIZE(x) (x + PAGE_SIZE)
#define ALIGN_ADDR(a) ((a + PAGE_SIZE) & ~(PAGE_SIZE - 1))

int main(int argc, char* argv[] ) {
    //hipError_t cu_ret;
    //hipDevice_t cu_device;
    //hipCtx_t cu_ctx;
    int ret = 0;
    unsigned gpu_id;
    unsigned orig_size;
    unsigned aligned_size;
    unsigned loc;
    hipError_t crt;
    //hipDeviceptr_t addr = 0;
    //hipDeviceptr_t aligned_addr = 0;
    void* addr;
    void* aligned_addr;
    const char* ctrl_name;
    struct bafs_dma_t dma_handle;

    struct bafs_ctrl_t ctrl_handle;

    if (argc < 4) {
        fprintf(stderr, "Please specify the memory size, controller, and gpu id.\n");
        exit(EXIT_FAILURE);
    }

    orig_size = strtoul(argv[1], NULL, 0);

    aligned_size = ALIGN_SIZE(orig_size);
    ctrl_name = argv[2];
    gpu_id = strtoul(argv[3], NULL, 0);
    loc = BAFS_MEM_CUDA;

    // cu_ret = hipInit(0);
    // if (cu_ret != hipSuccess) {
    //     perror("Unable to init cu driver");
    //     goto out_err;
    // }

    // cu_ret = hipDeviceGet(&cu_device, gpu_id);
    // if (cu_ret != hipSuccess) {
    //     perror("Unable to get cu device");
    //     goto out_err;
    // }

    // cu_ret = hipCtxCreate(&cu_ctx, 0, cu_device);
    // if (cu_ret != hipSuccess) {
    //     perror("Unable to create cu ctx");
    //     goto out_err;
    // }

    /* cu_ret = hipMalloc(&addr, aligned_size); */
    /* if (cu_ret != hipSuccess) { */
    /*     perror("Unable to create cu ctx"); */
    /*     goto out_destroy_ctx; */
    /* } */

    crt = hipMalloc(&addr, aligned_size);
    if (crt != hipSuccess) {
        perror("hipMalloc failed\n");
        goto out_destroy_ctx;
    }

    aligned_addr = addr;//(void*)ALIGN_ADDR(((unsigned long)addr));

    printf("orig_addr: %llx\taligned_addr: %llx\n", addr, aligned_addr);


    ret = bafs_core_map((void**)&aligned_addr, orig_size, loc);

    if (ret) {
        perror("Error while pinning memory");
        goto out_free_mem;
    }

    printf("Successfully registered and pinned memory\n");


    ret = bafs_ctrl_open(ctrl_name, &ctrl_handle);
    if (ret) {
        perror("Error while openning ctrl");
        goto out_free_mem;
    }

    printf("Successfully registered and pinned memory\n");

    dma_handle.dma_addrs = (void**)malloc(sizeof(void*) * ((orig_size + PAGE_SIZE - 1) / PAGE_SIZE));
    if (dma_handle.dma_addrs == NULL) {
        perror("Error allocating dma addresses");
        goto out_free_mem;
    }

    ret = bafs_ctrl_dma_map_mem((void*)aligned_addr, &dma_handle, &ctrl_handle);
    if (ret) {
        perror("Error while dma mapping memory");
        goto out_free_mem;
    }

    printf("Successfully dma mapped cpu memory\n");



    return EXIT_SUCCESS;

out_free_mem:
    hipFree (addr);
out_destroy_ctx:
    //hipCtxDestroy(cu_ctx);
out_err:
    exit(EXIT_FAILURE);

}
